#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Alfo_RotateNN.hpp>
#include <assert.h>
#include <iostream>
using namespace std;

void __global__ kernel_add_one(int* a, int length) {
    int gid = threadIdx.x + blockDim.x*blockIdx.x;

    while(gid < length) {
    	a[gid] += 1;
        gid += blockDim.x*gridDim.x;
    }
}


GPUAdder::GPUAdder (int* array_host_, int length_) {
  array_host = array_host_;
  length = length_;
  int size = length * sizeof(int);
  hipError_t err = hipMalloc((void**) &array_device, size);
  assert(err == 0);
  err = hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
  assert(err == 0);
}

void GPUAdder::increment() {
  kernel_add_one<<<64, 64>>>(array_device, length);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void GPUAdder::retreive() {
  int size = length * sizeof(int);
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0) { cout << err << endl; assert(0); }
}

void GPUAdder::retreive_to (int* array_host_, int length_) {
  assert(length == length_);
  int size = length * sizeof(int);
  hipMemcpy(array_host_, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

GPUAdder::~GPUAdder() {
  hipFree(array_device);
}